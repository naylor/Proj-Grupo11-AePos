#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "host.cuh"
#include "kernel.cuh"

#define BLOCK_DIM 32
#define BLOCK_DEFAULT 512


texture<unsigned char, hipTextureType2D> tex8u;



float box_filter_8u_c1(initialParams* ct, PPMImageParams* imageParams, PPMThread* thread, int numThread, hipStream_t* streamSmooth)
{

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int linhasIn = thread[numThread].linhasIn;
    double linhasOut = thread[numThread].linhasOut;

    const int width = imageParams->coluna;
    const int height = (thread[numThread].lf-thread[numThread].li)+1;
    const int widthStep = imageParams->coluna;

    unsigned char CPUinput[linhasIn];
    unsigned char CPUoutput[width*height];

    for(int t=0; t<linhasIn; t++)
        CPUinput[t] = thread[numThread].pgmIn[t].gray;

    //Declare GPU pointer
    unsigned char *GPU_input, *GPU_output;

    //Allocate 2D memory on GPU. Also known as Pitch Linear Memory
    size_t gpu_image_pitch = 0;
    hipMallocPitch<unsigned char>(&GPU_input,&gpu_image_pitch,width,thread[numThread].linhas);
    hipMallocPitch<unsigned char>(&GPU_output,&gpu_image_pitch,width,height);

    //Copy data from host to device.
    hipMemcpy2DAsync(GPU_input,gpu_image_pitch,CPUinput,widthStep,width,thread[numThread].linhas,hipMemcpyHostToDevice, streamSmooth[numThread]);

    //Bind the image to the texture. Now the kernel will read the input image through the texture cache.
    //Use tex2D function to read the image
    hipBindTexture2D(NULL,tex8u,GPU_input,width,thread[numThread].linhas,gpu_image_pitch);

    /*
     * Set the behavior of tex2D for out-of-range image reads.
     * hipAddressModeBorder = Read Zero
     * hipAddressModeClamp  = Read the nearest border pixel
     * We can skip this step. The default mode is Clamp.
     */
    //tex8u.addressMode[0] = tex8u.addressMode[1] = hipAddressModeBorder;

    /*
     * Specify a block size. 256 threads per block are sufficient.
     * It can be increased, but keep in mind the limitations of the GPU.
     * Older GPUs allow maximum 512 threads per block.
     * Current GPUs allow maximum 1024 threads per block
     */

    dim3 block_size(16,16);

    /*
     * Specify the grid size for the GPU.
     * Make it generalized, so that the size of grid changes according to the input image size
     */

    dim3 grid_size;
    grid_size.x = (width + block_size.x - 1)/block_size.x;  /*< Greater than or equal to image width */
    grid_size.y = (height + block_size.y - 1)/block_size.y; /*< Greater than or equal to image height */

    hipEventRecord(start, 0);
    box_filter_kernel_8u_c1<<<grid_size,block_size, 0, streamSmooth[numThread]>>>(GPU_output,width,imageParams->linha,gpu_image_pitch,thread[numThread].lf,thread[numThread].li);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    //Copy the results back to CPU
    hipMemcpy2DAsync(CPUoutput,widthStep,GPU_output,gpu_image_pitch,width,height,hipMemcpyDeviceToHost, streamSmooth[numThread]);

    for(int t=0; t<width*height; t++)
        thread[numThread].pgmOut[t].gray = CPUoutput[t];

    //Release the texture
    hipUnbindTexture(tex8u);

    //Free GPU memory
    hipFree(GPU_input);
    hipFree(GPU_output);

    hipEventElapsedTime(&time, start, stop);

    return time;
}

// FUNCAO __HOST__
// DEFINICAO DOS PARAMETROS DE CHAMADA DO KERNEL
void applySmooth(initialParams* ct, PPMImageParams* imageParams, PPMThread* block, int numBlock, hipStream_t* streamSmooth) {

    // DEFINE A QUANTIDADE DE LINHAS DO
    // BLOCO LIDO E DO BLOCO QUE SERA
    // GRAVADO EM DISCO
    double linhasIn = block[numBlock].linhasIn;
    double linhasOut = block[numBlock].linhasOut;

    // SE A IMAGEM FOR PPM
    if (strcmp(imageParams->tipo, "P6")==0) {
        // VARIAVEL PARA COPIA DA IMAGEM
        // PARA O KERNEL
        PPMPixel* kInput;
        PPMPixel* kOutput;

        // ALOCAR MEMORIA
        hipMalloc( (void**) &kInput, linhasIn);
        hipMalloc( (void**) &kOutput, linhasOut);

        // DEFINICAO DO TAMANHO PADRAO
        // DO BLOCO
        dim3 blockDims(BLOCK_DEFAULT,1,1);
        // SE A OPCAO DE SHARED MEMORY
        // FOR ATIVADA, DEFINE O TAMANHO
        // DO BLOCO PARA 32
        if (ct->sharedMemory == 1)
            blockDims.x = BLOCK_DIM;
        dim3 gridDims((unsigned int) ceil((double)(linhasIn/blockDims.x)), 1, 1 );

        // EXECUTA O CUDAMEMCPY
        // ASSINCRONO OU SINCRONO
        if (ct->async == 1)
            hipMemcpyAsync( kInput, block[numBlock].ppmIn, linhasIn, hipMemcpyHostToDevice, streamSmooth[numBlock] );
        else
            hipMemcpy( kInput, block[numBlock].ppmIn, linhasIn, hipMemcpyHostToDevice);

        // EXECUTA A FUNCAO SMOOTH NO KERNEL
        // SE A OPCAO DE SHARED MEMORY FOR ATIVADA
        // CHAMA A FUNCAO smoothPPM_SH
        if (ct->async == 1) {
            if (ct->sharedMemory == 1)
                smoothPPM_SH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPPM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        } else {
            if (ct->sharedMemory == 1)
                smoothPPM_SH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPPM_noSH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        }

        // RETORNA A IMAGEM PARA
        // A VARIAVEL DE SAIDA PARA
        // GRAVACAO NO ARQUIVO
        if (ct->async == 1)
            hipMemcpyAsync(block[numBlock].ppmOut, kOutput, linhasOut, hipMemcpyDeviceToHost, streamSmooth[numBlock] );
        else
            hipMemcpy(block[numBlock].ppmOut, kOutput, linhasOut, hipMemcpyDeviceToHost );

        // LIBERA A MEMORIA
        hipFree(kInput);
        hipFree(kOutput);
    }

    // SE A IMAGEM FOR PPM
    if (strcmp(imageParams->tipo, "P5")==0) {
        // VARIAVEL PARA COPIA DA IMAGEM
        // PARA O KERNEL
        PGMPixel* kInput;
        PGMPixel* kOutput;

        // ALOCAR MEMORIA
        hipMalloc( (void**) &kInput, linhasIn);
        hipMalloc( (void**) &kOutput, linhasOut);

        // DEFINICAO DO TAMANHO PADRAO
        // DO BLOCO
        dim3 blockDims(BLOCK_DEFAULT,1,1);
        // SE A OPCAO DE SHARED MEMORY
        // FOR ATIVADA, DEFINE O TAMANHO
        // DO BLOCO PARA 32
        if (ct->sharedMemory == 1)
            blockDims.x = BLOCK_DIM;
        dim3 gridDims((unsigned int) ceil((double)(linhasIn/blockDims.x)), 1, 1 );

        // EXECUTA O CUDAMEMCPY
        // ASSINCRONO OU SINCRONO
        if (ct->async == 1)
            hipMemcpyAsync( kInput, block[numBlock].pgmIn, linhasIn, hipMemcpyHostToDevice, streamSmooth[numBlock] );
        else
            hipMemcpy( kInput, block[numBlock].pgmIn, linhasIn, hipMemcpyHostToDevice);

        // EXECUTA A FUNCAO SMOOTH NO KERNEL
        // SE A OPCAO DE SHARED MEMORY FOR ATIVADA
        // CHAMA A FUNCAO smoothPPM_SH
        if (ct->async == 1) {
            if (ct->sharedMemory == 1)
                smoothPGM_SH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPGM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        } else {
            if (ct->sharedMemory == 1)
                smoothPGM_SH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPGM_noSH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        }

        // RETORNA A IMAGEM PARA
        // A VARIAVEL DE SAIDA PARA
        // GRAVACAO NO ARQUIVO
        if (ct->async == 1)
            hipMemcpyAsync(block[numBlock].pgmOut, kOutput, linhasOut, hipMemcpyDeviceToHost, streamSmooth[numBlock] );
        else
            hipMemcpy(block[numBlock].pgmOut, kOutput, linhasOut, hipMemcpyDeviceToHost );

        // LIBERA A MEMORIA
        hipFree(kInput);
        hipFree(kOutput);
    }

    hipDeviceSynchronize();

    if (ct->debug >= 1)
        printf("Apply Smooth[%d][%s] - li:%d, lf:%d\n",
               numBlock, imageParams->tipo, block[numBlock].li, block[numBlock].lf);

}

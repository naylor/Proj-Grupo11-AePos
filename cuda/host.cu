#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "host.cuh"
#include "kernel.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


float box_filter_8u_c1(initialParams* ct, PPMImageParams* imageParams,
                       PPMThread* thread, int numThread, hipStream_t* streamSmooth, int filtro)
{

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int linhasIn = thread[numThread].linhasIn;

    const int width = imageParams->coluna;
    const int height = (thread[numThread].lf-thread[numThread].li)+1;
    const int widthStep = imageParams->coluna;


    unsigned char* CPUinput;
    unsigned char* CPUoutput;
    CPUinput = (unsigned char *)malloc(linhasIn * sizeof(unsigned char));
        exit(1);

    CPUoutput = (unsigned char *)malloc(width*height * sizeof(unsigned char));


    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].red;
        if (filtro == 2)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].green;
        if (filtro == 3)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].blue;
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<linhasIn; t++)
            CPUinput[t] = thread[numThread].pgmIn[t].gray;
    }


    //Declare GPU pointer
    unsigned char *GPU_input, *GPU_output;

    //Allocate 2D memory on GPU. Also known as Pitch Linear Memory
    size_t gpu_image_pitch = 0;
    gpuErrchk( hipMallocPitch<unsigned char>(&GPU_input,&gpu_image_pitch,width,thread[numThread].linhas) );
    gpuErrchk( hipMallocPitch<unsigned char>(&GPU_output,&gpu_image_pitch,width,height) );

    //Copy data from host to device.
    gpuErrchk( hipMemcpy2DAsync(GPU_input,gpu_image_pitch,CPUinput,widthStep,width,thread[numThread].linhas,hipMemcpyHostToDevice, streamSmooth[numThread]) );

    //Bind the image to the texture. Now the kernel will read the input image through the texture cache.
    //Use tex2D function to read the image
   gpuErrchk( hipBindTexture2D(NULL,tex8u,GPU_input,width,thread[numThread].linhas,gpu_image_pitch) );

    /*
     * Set the behavior of tex2D for out-of-range image reads.
     * hipAddressModeBorder = Read Zero
     * hipAddressModeClamp  = Read the nearest border pixel
     * We can skip this step. The default mode is Clamp.
     */
    //tex8u.addressMode[0] = tex8u.addressMode[1] = hipAddressModeBorder;

    /*
     * Specify a block size. 256 threads per block are sufficient.
     * It can be increased, but keep in mind the limitations of the GPU.
     * Older GPUs allow maximum 512 threads per block.
     * Current GPUs allow maximum 1024 threads per block
     */

    dim3 block_size(16,16);

    /*
     * Specify the grid size for the GPU.
     * Make it generalized, so that the size of grid changes according to the input image size
     */

    dim3 grid_size;
    grid_size.x = (width + block_size.x - 1)/block_size.x;  /*< Greater than or equal to image width */
    grid_size.y = (height + block_size.y - 1)/block_size.y; /*< Greater than or equal to image height */

    hipEventRecord(start, 0);
    box_filter_kernel_8u_c1<<<grid_size,block_size, 0, streamSmooth[numThread]>>>(GPU_output,width,imageParams->linha,gpu_image_pitch,thread[numThread].lf,thread[numThread].li);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    //Copy the results back to CPU
    hipMemcpy2DAsync(CPUoutput,widthStep,GPU_output,gpu_image_pitch,width,height,hipMemcpyDeviceToHost, streamSmooth[numThread]);

    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].red = CPUoutput[t];
        if (filtro == 2)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].green = CPUoutput[t];
        if (filtro == 3)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].blue = CPUoutput[t];
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<width*height; t++)
            thread[numThread].pgmOut[t].gray = CPUoutput[t];
    }

    //Release the texture
    hipUnbindTexture(tex8u);

    //Free GPU memory
    hipFree(GPU_input);
    hipFree(GPU_output);

    hipEventElapsedTime(&time, start, stop);

    return time;
}

// FUNCAO __HOST__
// DEFINICAO DOS PARAMETROS DE CHAMADA DO KERNEL
void applySmooth(initialParams* ct, PPMImageParams* imageParams, PPMThread* block, int numBlock, hipStream_t* streamSmooth) {

    // DEFINE A QUANTIDADE DE LINHAS DO
    // BLOCO LIDO E DO BLOCO QUE SERA
    // GRAVADO EM DISCO
    double linhasIn = block[numBlock].linhasIn;
    double linhasOut = block[numBlock].linhasOut;

    // SE A IMAGEM FOR PPM
    if (strcmp(imageParams->tipo, "P6")==0) {
        // VARIAVEL PARA COPIA DA IMAGEM
        // PARA O KERNEL
        PPMPixel* kInput;
        PPMPixel* kOutput;

        // ALOCAR MEMORIA
        hipMalloc( (void**) &kInput, linhasIn);
        hipMalloc( (void**) &kOutput, linhasOut);

        // DEFINICAO DO TAMANHO PADRAO
        // DO BLOCO
        dim3 blockDims(512,1,1);
        dim3 gridDims((unsigned int) ceil((double)(linhasIn/blockDims.x)), 1, 1 );

        // EXECUTA O CUDAMEMCPY
        // ASSINCRONO
        hipMemcpyAsync( kInput, block[numBlock].ppmIn, linhasIn, hipMemcpyHostToDevice, streamSmooth[numBlock] );

        // EXECUTA A FUNCAO SMOOTH NO KERNEL
        smoothPPM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);

        // RETORNA A IMAGEM PARA
        // A VARIAVEL DE SAIDA PARA
        // GRAVACAO NO ARQUIVO
        hipMemcpyAsync(block[numBlock].ppmOut, kOutput, linhasOut, hipMemcpyDeviceToHost, streamSmooth[numBlock] );

        // LIBERA A MEMORIA
        hipFree(kInput);
        hipFree(kOutput);
    }

    // SE A IMAGEM FOR PPM
    if (strcmp(imageParams->tipo, "P5")==0) {
        // VARIAVEL PARA COPIA DA IMAGEM
        // PARA O KERNEL
        PGMPixel* kInput;
        PGMPixel* kOutput;

        // ALOCAR MEMORIA
        hipMalloc( (void**) &kInput, linhasIn);
        hipMalloc( (void**) &kOutput, linhasOut);

        // DEFINICAO DO TAMANHO PADRAO
        // DO BLOCO
        dim3 blockDims(512,1,1);
        dim3 gridDims((unsigned int) ceil((double)(linhasIn/blockDims.x)), 1, 1 );

        // EXECUTA O CUDAMEMCPY
        hipMemcpyAsync( kInput, block[numBlock].pgmIn, linhasIn, hipMemcpyHostToDevice, streamSmooth[numBlock] );

        // EXECUTA A FUNCAO SMOOTH NO KERNEL
        smoothPGM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);

        // RETORNA A IMAGEM PARA
        // A VARIAVEL DE SAIDA PARA
        // GRAVACAO NO ARQUIVO
        hipMemcpyAsync(block[numBlock].pgmOut, kOutput, linhasOut, hipMemcpyDeviceToHost, streamSmooth[numBlock] );

        // LIBERA A MEMORIA
        hipFree(kInput);
        hipFree(kOutput);
    }

    hipDeviceSynchronize();

    if (ct->debug >= 1)
        printf("Apply Smooth[%d][%s] - li:%d, lf:%d\n",
               numBlock, imageParams->tipo, block[numBlock].li, block[numBlock].lf);

}

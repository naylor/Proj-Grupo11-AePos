#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "host.cuh"
#include "kernel.cuh"

#define BLOCK_DIM 32
#define BLOCK_DEFAULT 512

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

texture<unsigned char, hipTextureType2D> tex8u;

//Box Filter Kernel For Gray scale image with 8bit depth
__global__ void box_filter_kernel_8u_c1(unsigned char* output,const int width, const int height, const size_t pitch, const int lf, const int li)
{

    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;


    float output_value = 0.0f;
    int cont = 0;

    int c = xIndex % width; // COLUNA
    int l = (xIndex-c)/width; // LINHA


    // TIRANDO A BORDA DO PROCESSAMENTO
    if ( l > lf-li || c < 2 || c > width-2 || (li == 0 && l < 2) || (lf==height-1 && l > (lf-li)-2) )
        return;

    int inicio = 0;
    if (li != 0)
        inicio = 2;

        //Sum the window pixels
        for(int l2= -2; l2<=2; l2++)
        {
            for(int c2=-2; c2<=2; c2++)
            {
            if(l2 >= 0 && c2 >= 0) {


                //No need to worry about Out-Of-Range access. tex2D automatically handles it.
                output_value += tex2D(tex8u,inicio+ xIndex+l2,yIndex + c2);
                cont++;
            }
            }
        }

        //Average the output value
        output_value = output_value/cont;

        //Write the averaged value to the output.
        //Transform 2D index to 1D index, because image is actually in linear memory
        int index = yIndex * pitch + xIndex;

        output[index] = static_cast<unsigned char>(output_value);

}


float box_filter_8u_c1(initialParams* ct, PPMImageParams* imageParams,
                       PPMThread* thread, int numThread, hipStream_t* streamSmooth, int filtro)
{

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int linhasIn = thread[numThread].linhasIn;
    double linhasOut = thread[numThread].linhasOut;

    const int width = imageParams->coluna;
    const int height = (thread[numThread].lf-thread[numThread].li)+1;
    const int widthStep = imageParams->coluna;

    unsigned char CPUinput[linhasIn];
    unsigned char CPUoutput[width*height];

    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].red;
        if (filtro == 2)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].green;
        if (filtro == 3)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].blue;
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<linhasIn; t++)
            CPUinput[t] = thread[numThread].pgmIn[t].gray;
    }

    //Declare GPU pointer
    unsigned char *GPU_input, *GPU_output;

    //Allocate 2D memory on GPU. Also known as Pitch Linear Memory
    size_t gpu_image_pitch = 0;
    gpuErrchk( hipMallocPitch<unsigned char>(&GPU_input,&gpu_image_pitch,width,thread[numThread].linhas) );
    gpuErrchk( hipMallocPitch<unsigned char>(&GPU_output,&gpu_image_pitch,width,height) );

    //Copy data from host to device.
    gpuErrchk( hipMemcpy2D(GPU_input,gpu_image_pitch,CPUinput,widthStep,width,thread[numThread].linhas,hipMemcpyHostToDevice) );

    //Bind the image to the texture. Now the kernel will read the input image through the texture cache.
    //Use tex2D function to read the image
   gpuErrchk( hipBindTexture2D(NULL,tex8u,GPU_input,width,thread[numThread].linhas,gpu_image_pitch) );

    /*
     * Set the behavior of tex2D for out-of-range image reads.
     * hipAddressModeBorder = Read Zero
     * hipAddressModeClamp  = Read the nearest border pixel
     * We can skip this step. The default mode is Clamp.
     */
    //tex8u.addressMode[0] = tex8u.addressMode[1] = hipAddressModeBorder;

    /*
     * Specify a block size. 256 threads per block are sufficient.
     * It can be increased, but keep in mind the limitations of the GPU.
     * Older GPUs allow maximum 512 threads per block.
     * Current GPUs allow maximum 1024 threads per block
     */

    dim3 block_size(16,16);

    /*
     * Specify the grid size for the GPU.
     * Make it generalized, so that the size of grid changes according to the input image size
     */

    dim3 grid_size;
    grid_size.x = (width + block_size.x - 1)/block_size.x;  /*< Greater than or equal to image width */
    grid_size.y = (height + block_size.y - 1)/block_size.y; /*< Greater than or equal to image height */

    hipEventRecord(start, 0);
    box_filter_kernel_8u_c1<<<grid_size,block_size>>>(GPU_output,width,imageParams->linha,gpu_image_pitch,thread[numThread].lf,thread[numThread].li);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    //Copy the results back to CPU
    hipMemcpy2D(CPUoutput,widthStep,GPU_output,gpu_image_pitch,width,height,hipMemcpyDeviceToHost);

    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].red = CPUoutput[t];
        if (filtro == 2)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].green = CPUoutput[t];
        if (filtro == 3)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].blue = CPUoutput[t];
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<width*height; t++)
            thread[numThread].pgmOut[t].gray = CPUoutput[t];
    }

    //Release the texture
    hipUnbindTexture(tex8u);

    //Free GPU memory
    hipFree(GPU_input);
    hipFree(GPU_output);

    hipEventElapsedTime(&time, start, stop);

    return time;
}

// FUNCAO __HOST__
// DEFINICAO DOS PARAMETROS DE CHAMADA DO KERNEL
void applySmooth(initialParams* ct, PPMImageParams* imageParams, PPMThread* block, int numBlock, hipStream_t* streamSmooth) {

    // DEFINE A QUANTIDADE DE LINHAS DO
    // BLOCO LIDO E DO BLOCO QUE SERA
    // GRAVADO EM DISCO
    double linhasIn = block[numBlock].linhasIn;
    double linhasOut = block[numBlock].linhasOut;

    // SE A IMAGEM FOR PPM
    if (strcmp(imageParams->tipo, "P6")==0) {
        // VARIAVEL PARA COPIA DA IMAGEM
        // PARA O KERNEL
        PPMPixel* kInput;
        PPMPixel* kOutput;

        // ALOCAR MEMORIA
        hipMalloc( (void**) &kInput, linhasIn);
        hipMalloc( (void**) &kOutput, linhasOut);

        // DEFINICAO DO TAMANHO PADRAO
        // DO BLOCO
        dim3 blockDims(BLOCK_DEFAULT,1,1);
        // SE A OPCAO DE SHARED MEMORY
        // FOR ATIVADA, DEFINE O TAMANHO
        // DO BLOCO PARA 32
        if (ct->sharedMemory == 1)
            blockDims.x = BLOCK_DIM;
        dim3 gridDims((unsigned int) ceil((double)(linhasIn/blockDims.x)), 1, 1 );

        // EXECUTA O CUDAMEMCPY
        // ASSINCRONO OU SINCRONO
        if (ct->async == 1)
            hipMemcpyAsync( kInput, block[numBlock].ppmIn, linhasIn, hipMemcpyHostToDevice, streamSmooth[numBlock] );
        else
            hipMemcpy( kInput, block[numBlock].ppmIn, linhasIn, hipMemcpyHostToDevice);

        // EXECUTA A FUNCAO SMOOTH NO KERNEL
        // SE A OPCAO DE SHARED MEMORY FOR ATIVADA
        // CHAMA A FUNCAO smoothPPM_SH
        if (ct->async == 1) {
            if (ct->sharedMemory == 1)
                smoothPPM_SH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPPM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        } else {
            if (ct->sharedMemory == 1)
                smoothPPM_SH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPPM_noSH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        }

        // RETORNA A IMAGEM PARA
        // A VARIAVEL DE SAIDA PARA
        // GRAVACAO NO ARQUIVO
        if (ct->async == 1)
            hipMemcpyAsync(block[numBlock].ppmOut, kOutput, linhasOut, hipMemcpyDeviceToHost, streamSmooth[numBlock] );
        else
            hipMemcpy(block[numBlock].ppmOut, kOutput, linhasOut, hipMemcpyDeviceToHost );

        // LIBERA A MEMORIA
        hipFree(kInput);
        hipFree(kOutput);
    }

    // SE A IMAGEM FOR PPM
    if (strcmp(imageParams->tipo, "P5")==0) {
        // VARIAVEL PARA COPIA DA IMAGEM
        // PARA O KERNEL
        PGMPixel* kInput;
        PGMPixel* kOutput;

        // ALOCAR MEMORIA
        hipMalloc( (void**) &kInput, linhasIn);
        hipMalloc( (void**) &kOutput, linhasOut);

        // DEFINICAO DO TAMANHO PADRAO
        // DO BLOCO
        dim3 blockDims(BLOCK_DEFAULT,1,1);
        // SE A OPCAO DE SHARED MEMORY
        // FOR ATIVADA, DEFINE O TAMANHO
        // DO BLOCO PARA 32
        if (ct->sharedMemory == 1)
            blockDims.x = BLOCK_DIM;
        dim3 gridDims((unsigned int) ceil((double)(linhasIn/blockDims.x)), 1, 1 );

        // EXECUTA O CUDAMEMCPY
        // ASSINCRONO OU SINCRONO
        if (ct->async == 1)
            hipMemcpyAsync( kInput, block[numBlock].pgmIn, linhasIn, hipMemcpyHostToDevice, streamSmooth[numBlock] );
        else
            hipMemcpy( kInput, block[numBlock].pgmIn, linhasIn, hipMemcpyHostToDevice);

        // EXECUTA A FUNCAO SMOOTH NO KERNEL
        // SE A OPCAO DE SHARED MEMORY FOR ATIVADA
        // CHAMA A FUNCAO smoothPPM_SH
        if (ct->async == 1) {
            if (ct->sharedMemory == 1)
                smoothPGM_SH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPGM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        } else {
            if (ct->sharedMemory == 1)
                smoothPGM_SH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPGM_noSH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        }

        // RETORNA A IMAGEM PARA
        // A VARIAVEL DE SAIDA PARA
        // GRAVACAO NO ARQUIVO
        if (ct->async == 1)
            hipMemcpyAsync(block[numBlock].pgmOut, kOutput, linhasOut, hipMemcpyDeviceToHost, streamSmooth[numBlock] );
        else
            hipMemcpy(block[numBlock].pgmOut, kOutput, linhasOut, hipMemcpyDeviceToHost );

        // LIBERA A MEMORIA
        hipFree(kInput);
        hipFree(kOutput);
    }

    hipDeviceSynchronize();

    if (ct->debug >= 1)
        printf("Apply Smooth[%d][%s] - li:%d, lf:%d\n",
               numBlock, imageParams->tipo, block[numBlock].li, block[numBlock].lf);

}

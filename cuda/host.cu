#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "host.cuh"
#include "kernel.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

texture<unsigned char, hipTextureType2D> tex8u;


//Box Filter Kernel For Gray scale image with 8bit depth
__global__ void box_filter_kernel_8u_c1(unsigned char* output,const int width, const int height, const size_t pitch, const int lf, const int li)
{

    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;


    float output_value = 0.0f;
    int cont = 0;

    // TIRANDO A BORDA DO PROCESSAMENTO
    if ( yIndex > lf-li || xIndex < 2 || xIndex > width-2 || (li == 0 && yIndex < 2) || (lf==height-1 && yIndex > (lf-li)-2) )
        return;

    int inicio = 0;
    if (li != 0)
        inicio = 2;

        for(int l2= -2; l2<=2; l2++)
        {
            for(int c2=-2; c2<=2; c2++)
            {
            if(l2 >= 0 && c2 >= 0) {
                output_value += tex2D(tex8u,inicio+ xIndex+l2,yIndex + c2);
                cont++;
            }
            }
        }

        //Average the output value
        output_value = output_value/cont;

        //Write the averaged value to the output.
        //Transform 2D index to 1D index, because image is actually in linear memory
        int index = yIndex * pitch + xIndex;
        //printf("Smooth index:%d, xIndex:%d yIndex %d lf-li %d\n",index, xIndex, yIndex, lf-li);

        output[index] = static_cast<unsigned char>(output_value);

}

float box_filter_8u_c1(initialParams* ct, PPMImageParams* imageParams,
                       PPMThread* thread, int numThread, hipStream_t* streamSmooth, int filtro)
{

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int linhasIn = thread[numThread].linhasIn;
    int linhasOut = thread[numThread].linhasOut;

    const int width = imageParams->coluna;
    const int height = (thread[numThread].lf-thread[numThread].li)+1;
    const int widthStep = imageParams->coluna;


    unsigned char* CPUinput;
    unsigned char* CPUoutput;
    CPUinput = (unsigned char *)malloc(linhasIn * sizeof(unsigned char));
    CPUoutput = (unsigned char *)malloc(width*height * sizeof(unsigned char));



    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].red;
        if (filtro == 2)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].green;
        if (filtro == 3)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].blue;
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<linhasIn; t++)
            CPUinput[t] = thread[numThread].pgmIn[t].gray;
    }


    //Declare GPU pointer
    unsigned char *GPU_input, *GPU_output;

    //Allocate 2D memory on GPU. Also known as Pitch Linear Memory
    size_t gpu_image_pitch = 0;
    gpuErrchk( hipMallocPitch<unsigned char>(&GPU_input,&gpu_image_pitch,width,thread[numThread].linhas) );
    gpuErrchk( hipMallocPitch<unsigned char>(&GPU_output,&gpu_image_pitch,width,height) );

    //Copy data from host to device.
    gpuErrchk( hipMemcpy2DAsync(GPU_input,gpu_image_pitch,CPUinput,widthStep,width,thread[numThread].linhas,hipMemcpyHostToDevice, streamSmooth[numThread]) );

    //Bind the image to the texture. Now the kernel will read the input image through the texture cache.
    //Use tex2D function to read the image
   gpuErrchk( hipBindTexture2D(NULL,tex8u,GPU_input,width,thread[numThread].linhas,gpu_image_pitch) );

    /*
     * Set the behavior of tex2D for out-of-range image reads.
     * hipAddressModeBorder = Read Zero
     * hipAddressModeClamp  = Read the nearest border pixel
     * We can skip this step. The default mode is Clamp.
     */
    //tex8u.addressMode[0] = tex8u.addressMode[1] = hipAddressModeBorder;

    /*
     * Specify a block size. 256 threads per block are sufficient.
     * It can be increased, but keep in mind the limitations of the GPU.
     * Older GPUs allow maximum 512 threads per block.
     * Current GPUs allow maximum 1024 threads per block
     */

    dim3 block_size(16,16);

    /*
     * Specify the grid size for the GPU.
     * Make it generalized, so that the size of grid changes according to the input image size
     */

    dim3 grid_size;
    grid_size.x = (width + block_size.x - 1)/block_size.x;  /*< Greater than or equal to image width */
    grid_size.y = (height + block_size.y - 1)/block_size.y; /*< Greater than or equal to image height */

    hipEventRecord(start, 0);
    box_filter_kernel_8u_c1<<<grid_size,block_size, 0, streamSmooth[numThread]>>>(GPU_output,width,imageParams->linha,gpu_image_pitch,thread[numThread].lf,thread[numThread].li);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    //Copy the results back to CPU
    hipMemcpy2DAsync(CPUoutput,widthStep,GPU_output,gpu_image_pitch,width,height,hipMemcpyDeviceToHost, streamSmooth[numThread]);

    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].red = CPUoutput[t];
        if (filtro == 2)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].green = CPUoutput[t];
        if (filtro == 3)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].blue = CPUoutput[t];
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<width*height; t++)
            thread[numThread].pgmOut[t].gray = CPUoutput[t];
    }

    //Release the texture
    hipUnbindTexture(tex8u);

    //Free GPU memory
    hipFree(GPU_input);
    hipFree(GPU_output);

    hipEventElapsedTime(&time, start, stop);

    return time;
}



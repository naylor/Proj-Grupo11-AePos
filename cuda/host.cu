#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "host.cuh"
#include "kernel.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


float box_filter_8u_c1(initialParams* ct, PPMImageParams* imageParams,
                       PPMThread* thread, int numThread, hipStream_t* streamSmooth, int filtro)
{

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    int linhasIn = thread[numThread].linhasIn;
    int linhasOut = thread[numThread].linhasOut;

    const int width = imageParams->coluna;
    const int height = (thread[numThread].lf-thread[numThread].li)+1;
    const int widthStep = imageParams->coluna;


    unsigned char* CPUinput;
    unsigned char* CPUoutput;
    CPUinput = (unsigned char *)malloc(linhasIn * sizeof(unsigned char));
    CPUoutput = (unsigned char *)malloc(width*height * sizeof(unsigned char));



    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].red;
        if (filtro == 2)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].green;
        if (filtro == 3)
            for(int t=0; t<linhasIn; t++)
                CPUinput[t] = thread[numThread].ppmIn[t].blue;
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<linhasIn; t++)
            CPUinput[t] = thread[numThread].pgmIn[t].gray;
    }


    //Declare GPU pointer
    unsigned char *GPU_input, *GPU_output;

    //Allocate 2D memory on GPU. Also known as Pitch Linear Memory
    size_t gpu_image_pitch = 0;
    gpuErrchk( hipMallocPitch<unsigned char>(&GPU_input,&gpu_image_pitch,width,thread[numThread].linhas) );
    gpuErrchk( hipMallocPitch<unsigned char>(&GPU_output,&gpu_image_pitch,width,height) );

    //Copy data from host to device.
    gpuErrchk( hipMemcpy2DAsync(GPU_input,gpu_image_pitch,CPUinput,widthStep,width,thread[numThread].linhas,hipMemcpyHostToDevice, streamSmooth[numThread]) );

    //Bind the image to the texture. Now the kernel will read the input image through the texture cache.
    //Use tex2D function to read the image
   gpuErrchk( hipBindTexture2D(NULL,tex8u,GPU_input,width,thread[numThread].linhas,gpu_image_pitch) );

    /*
     * Set the behavior of tex2D for out-of-range image reads.
     * hipAddressModeBorder = Read Zero
     * hipAddressModeClamp  = Read the nearest border pixel
     * We can skip this step. The default mode is Clamp.
     */
    //tex8u.addressMode[0] = tex8u.addressMode[1] = hipAddressModeBorder;

    /*
     * Specify a block size. 256 threads per block are sufficient.
     * It can be increased, but keep in mind the limitations of the GPU.
     * Older GPUs allow maximum 512 threads per block.
     * Current GPUs allow maximum 1024 threads per block
     */

    dim3 block_size(16,16);

    /*
     * Specify the grid size for the GPU.
     * Make it generalized, so that the size of grid changes according to the input image size
     */

    dim3 grid_size;
    grid_size.x = (width + block_size.x - 1)/block_size.x;  /*< Greater than or equal to image width */
    grid_size.y = (height + block_size.y - 1)/block_size.y; /*< Greater than or equal to image height */

    hipEventRecord(start, 0);
    box_filter_kernel_8u_c1<<<grid_size,block_size, 0, streamSmooth[numThread]>>>(GPU_output,width,imageParams->linha,gpu_image_pitch,thread[numThread].lf,thread[numThread].li);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    //Copy the results back to CPU
    hipMemcpy2DAsync(CPUoutput,widthStep,GPU_output,gpu_image_pitch,width,height,hipMemcpyDeviceToHost, streamSmooth[numThread]);

    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].red = CPUoutput[t];
        if (filtro == 2)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].green = CPUoutput[t];
        if (filtro == 3)
            for(int t=0; t<width*height; t++)
                thread[numThread].ppmOut[t].blue = CPUoutput[t];
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<width*height; t++)
            thread[numThread].pgmOut[t].gray = CPUoutput[t];
    }

    //Release the texture
    hipUnbindTexture(tex8u);

    //Free GPU memory
    hipFree(GPU_input);
    hipFree(GPU_output);

    hipEventElapsedTime(&time, start, stop);

    return time;
}



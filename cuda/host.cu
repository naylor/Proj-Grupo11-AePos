#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "host.cuh"
#include "kernel.cuh"

#define BLOCK_DIM 32
#define BLOCK_DEFAULT 512

texture<unsigned char, hipTextureType2D> tex8u;


// FUNCAO PARA APLICAR SMOOTH
// COM SHARED MEMORY EM IMAGENS PPM
__global__ void box_filter_kernel_8u_c1(unsigned char* output,const int width, const int height, const size_t pitch, const int fWidth, const int fHeight)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    const int filter_offset_x = fWidth/2;
    const int filter_offset_y = fHeight/2;

    float output_value = 0.0f;

    //Make sure the current thread is inside the image bounds
    if(xIndex<width && yIndex<height)
    {
        //Sum the window pixels
        for(int i= -filter_offset_x; i<=filter_offset_x; i++)
        {
            for(int j=-filter_offset_y; j<=filter_offset_y; j++)
            {
                //No need to worry about Out-Of-Range access. tex2D automatically handles it.
                output_value += tex2D(tex8u,xIndex + i,yIndex + j);
            }
        }

        //Average the output value
        output_value /= (fWidth * fHeight);

        //Write the averaged value to the output.
        //Transform 2D index to 1D index, because image is actually in linear memory
        int index = yIndex * pitch + xIndex;

        output[index] = static_cast<unsigned char>(output_value);
    }
}

void box_filter_8u_c1(PPMImageParams* imageParams, PPMBlock* block, int numBlock)
{
        const int widthStep=2;
        const int filterWidth=5;
        const int filterHeight=5;
        unsigned char* CPUinput;
        unsigned char* CPUoutput ;
        const int width = imageParams->coluna;
        const int height = imageParams->linha;

            for(int t=0; t<imageParams->coluna*imageParams->linha; t++)
                   CPUinput[t] =  block[numBlock].pgmIn[t].gray;

        printf("Apply Smooth[%d][%s] - li:%d, lf:%d\n",
               numBlock, imageParams->tipo, block[numBlock].li, block[numBlock].lf);

    /*
     * 2D memory is allocated as strided linear memory on GPU.
     * The terminologies "Pitch", "WidthStep", and "Stride" are exactly the same thing.
     * It is the size of a row in bytes.
     * It is not necessary that width = widthStep.
     * Total bytes occupied by the image = widthStep x height.
     */

    //Declare GPU pointer
    unsigned char *GPU_input, *GPU_output;

    //Allocate 2D memory on GPU. Also known as Pitch Linear Memory
    size_t gpu_image_pitch = 0;
    hipMallocPitch<unsigned char>(&GPU_input,&gpu_image_pitch,width,height);
    hipMallocPitch<unsigned char>(&GPU_output,&gpu_image_pitch,width,height);

    //Copy data from host to device.
    hipMemcpy2D(GPU_input,gpu_image_pitch,CPUinput,widthStep,width,height,hipMemcpyHostToDevice);

    //Bind the image to the texture. Now the kernel will read the input image through the texture cache.
    //Use tex2D function to read the image
    hipBindTexture2D(NULL,tex8u,GPU_input,width,height,gpu_image_pitch);

    /*
     * Set the behavior of tex2D for out-of-range image reads.
     * hipAddressModeBorder = Read Zero
     * hipAddressModeClamp  = Read the nearest border pixel
     * We can skip this step. The default mode is Clamp.
     */
    tex8u.addressMode[0] = tex8u.addressMode[1] = hipAddressModeBorder;

    /*
     * Specify a block size. 256 threads per block are sufficient.
     * It can be increased, but keep in mind the limitations of the GPU.
     * Older GPUs allow maximum 512 threads per block.
     * Current GPUs allow maximum 1024 threads per block
     */

    dim3 block_size(16,16);

    /*
     * Specify the grid size for the GPU.
     * Make it generalized, so that the size of grid changes according to the input image size
     */

    dim3 grid_size;
    grid_size.x = (width + block_size.x - 1)/block_size.x;  /*< Greater than or equal to image width */
    grid_size.y = (height + block_size.y - 1)/block_size.y; /*< Greater than or equal to image height */

    //Launch the kernel
    box_filter_kernel_8u_c1<<<grid_size,block_size>>>(GPU_output,width,height,gpu_image_pitch,filterWidth,filterHeight);

    //Copy the results back to CPU
    hipMemcpy2D(CPUoutput,widthStep,GPU_output,gpu_image_pitch,width,height,hipMemcpyDeviceToHost);

            for(int t=0; t<imageParams->coluna*imageParams->linha; t++)
                   block[numBlock].pgmIn[t].gray = CPUoutput[t];

    //Release the texture
    hipUnbindTexture(tex8u);

    //Free GPU memory
    hipFree(GPU_input);
    hipFree(GPU_output);
}

// FUNCAO __HOST__
// DEFINICAO DOS PARAMETROS DE CHAMADA DO KERNEL
void applySmooth(initialParams* ct, PPMImageParams* imageParams, PPMBlock* block, int numBlock, hipStream_t* streamSmooth) {

    // DEFINE A QUANTIDADE DE LINHAS DO
    // BLOCO LIDO E DO BLOCO QUE SERA
    // GRAVADO EM DISCO
    double linhasIn = block[numBlock].linhasIn;
    double linhasOut = block[numBlock].linhasOut;

    // SE A IMAGEM FOR PPM
    if (strcmp(imageParams->tipo, "P6")==0) {
        // VARIAVEL PARA COPIA DA IMAGEM
        // PARA O KERNEL
        PPMPixel* kInput;
        PPMPixel* kOutput;

        // ALOCAR MEMORIA
        hipMalloc( (void**) &kInput, linhasIn);
        hipMalloc( (void**) &kOutput, linhasOut);

        // DEFINICAO DO TAMANHO PADRAO
        // DO BLOCO
        dim3 blockDims(BLOCK_DEFAULT,1,1);
        // SE A OPCAO DE SHARED MEMORY
        // FOR ATIVADA, DEFINE O TAMANHO
        // DO BLOCO PARA 32
        //if (ct->sharedMemory == 1)
        //    blockDims.x = BLOCK_DIM;
        dim3 gridDims((unsigned int) ceil((double)(linhasIn/blockDims.x)), 1, 1 );

            //Declare GPU pointer
            unsigned char *GPU_input, *GPU_output;
            //Allocate 2D memory on GPU. Also known as Pitch Linear Memory
            size_t gpu_image_pitch = 0;

        dim3 block_size(16,16);
dim3 grid_size;
    grid_size.x = (imageParams->coluna + block_size.x - 1)/block_size.x;  /*< Greater than or equal to image width */
    grid_size.y = (imageParams->linha + block_size.y - 1)/block_size.y; /*< Greater than or equal to image height */

        // EXECUTA O CUDAMEMCPY
        // ASSINCRONO OU SINCRONO
        if (ct->async == 1) {

            hipMallocPitch<unsigned char>(&GPU_input,&gpu_image_pitch,imageParams->coluna,imageParams->linha);
            hipMallocPitch<unsigned char>(&GPU_output,&gpu_image_pitch,imageParams->coluna,imageParams->linha);

            //Copy data from host to device.
            hipMemcpy2D(GPU_input,gpu_image_pitch,block[numBlock].ppmIn,imageParams->coluna,imageParams->coluna,imageParams->linha,hipMemcpyHostToDevice);

            //Bind the image to the texture. Now the kernel will read the input image through the texture cache.
            //Use tex2D function to read the image
            hipBindTexture2D(NULL,tex8u,GPU_input,imageParams->coluna,imageParams->linha,gpu_image_pitch);

            /*
             * Set the behavior of tex2D for out-of-range image reads.
             * hipAddressModeBorder = Read Zero
             * hipAddressModeClamp  = Read the nearest border pixel
             * We can skip this step. The default mode is Clamp.
             */
            tex8u.addressMode[0] = tex8u.addressMode[1] = hipAddressModeBorder;
        } else
            hipMemcpy( kInput, block[numBlock].ppmIn, linhasIn, hipMemcpyHostToDevice);

        // EXECUTA A FUNCAO SMOOTH NO KERNEL
        // SE A OPCAO DE SHARED MEMORY FOR ATIVADA
        // CHAMA A FUNCAO smoothPPM_SH
        if (ct->async == 1) {
            if (ct->sharedMemory == 1)
                smoothPPM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPPM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        } else {
            if (ct->sharedMemory == 1)
                smoothPPM_noSH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPPM_noSH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        }

        // RETORNA A IMAGEM PARA
        // A VARIAVEL DE SAIDA PARA
        // GRAVACAO NO ARQUIVO
        if (ct->async == 1) {
            //Copy the results back to CPU
            hipMemcpy2D(block[numBlock].ppmOut,imageParams->coluna,GPU_output,gpu_image_pitch,imageParams->coluna,imageParams->linha,hipMemcpyDeviceToHost);

            //Release the texture
            hipUnbindTexture(tex8u);
        } else
            hipMemcpy(block[numBlock].ppmOut, kOutput, linhasOut, hipMemcpyDeviceToHost );

        // LIBERA A MEMORIA
        hipFree(kInput);
        hipFree(kOutput);
    }

    // SE A IMAGEM FOR PPM
    if (strcmp(imageParams->tipo, "P5")==0) {
        // VARIAVEL PARA COPIA DA IMAGEM
        // PARA O KERNEL
        PGMPixel* kInput;
        PGMPixel* kOutput;

        // ALOCAR MEMORIA
        hipMalloc( (void**) &kInput, linhasIn);
        hipMalloc( (void**) &kOutput, linhasOut);

        // DEFINICAO DO TAMANHO PADRAO
        // DO BLOCO
        dim3 blockDims(BLOCK_DEFAULT,1,1);
        // SE A OPCAO DE SHARED MEMORY
        // FOR ATIVADA, DEFINE O TAMANHO
        // DO BLOCO PARA 32
        if (ct->sharedMemory == 1)
            blockDims.x = BLOCK_DIM;
        dim3 gridDims((unsigned int) ceil((double)(linhasIn/blockDims.x)), 1, 1 );

        // EXECUTA O CUDAMEMCPY
        // ASSINCRONO OU SINCRONO
        if (ct->async == 1)
            hipMemcpyAsync( kInput, block[numBlock].pgmIn, linhasIn, hipMemcpyHostToDevice, streamSmooth[numBlock] );
        else
            hipMemcpy( kInput, block[numBlock].pgmIn, linhasIn, hipMemcpyHostToDevice);

        // EXECUTA A FUNCAO SMOOTH NO KERNEL
        // SE A OPCAO DE SHARED MEMORY FOR ATIVADA
        // CHAMA A FUNCAO smoothPPM_SH
        if (ct->async == 1) {
            if (ct->sharedMemory == 1)
                smoothPGM_SH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPGM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        } else {
            if (ct->sharedMemory == 1)
                smoothPGM_SH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPGM_noSH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        }

        // RETORNA A IMAGEM PARA
        // A VARIAVEL DE SAIDA PARA
        // GRAVACAO NO ARQUIVO
        if (ct->async == 1)
            hipMemcpyAsync(block[numBlock].pgmOut, kOutput, linhasOut, hipMemcpyDeviceToHost, streamSmooth[numBlock] );
        else
            hipMemcpy(block[numBlock].pgmOut, kOutput, linhasOut, hipMemcpyDeviceToHost );

        // LIBERA A MEMORIA
        hipFree(kInput);
        hipFree(kOutput);
    }

    hipDeviceSynchronize();

    if (ct->debug >= 1)
        printf("Apply Smooth[%d][%s] - li:%d, lf:%d\n",
               numBlock, imageParams->tipo, block[numBlock].li, block[numBlock].lf);

}

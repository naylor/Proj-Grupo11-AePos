#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "host.cuh"
#include "kernel.cuh"

#define BLOCK_DIM 32
#define BLOCK_DEFAULT 512

texture<unsigned char, hipTextureType2D> tex8u;


// FUNCAO PARA APLICAR SMOOTH
// COM SHARED MEMORY EM IMAGENS PPM
__global__ void smoothPPM_SH(PPMPixel* kInput, PPMPixel* kOutput, int coluna, int linha, int li, int lf) {
    // OFFSET DA COLUNA*LINHA
    unsigned int offset = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int yIndex = blockIdx.y * blockDim.y + threadIdx.y;

    int c = offset % coluna; // COLUNA
    int l = (offset-c)/coluna; // LINHA

    // TIRANDO A BORDA DO PROCESSAMENTO
    if ( l > lf-li || c < 2 || c > coluna-2 || (li == 0 && l < 2) || (lf==linha-1 && l > (lf-li)-2) )
        return;


        kOutput[offset].blue = tex2D(tex8u,offset,yIndex);



}

// FUNCAO __HOST__
// DEFINICAO DOS PARAMETROS DE CHAMADA DO KERNEL
void applySmooth(initialParams* ct, PPMImageParams* imageParams, PPMBlock* block, int numBlock, hipStream_t* streamSmooth) {

    // DEFINE A QUANTIDADE DE LINHAS DO
    // BLOCO LIDO E DO BLOCO QUE SERA
    // GRAVADO EM DISCO
    double linhasIn = block[numBlock].linhasIn;
    double linhasOut = block[numBlock].linhasOut;

    // SE A IMAGEM FOR PPM
    if (strcmp(imageParams->tipo, "P6")==0) {
        // VARIAVEL PARA COPIA DA IMAGEM
        // PARA O KERNEL
        PPMPixel* kInput;
        PPMPixel* kOutput;

        // ALOCAR MEMORIA
        hipMalloc( (void**) &kInput, linhasIn);
        hipMalloc( (void**) &kOutput, linhasOut);

        // DEFINICAO DO TAMANHO PADRAO
        // DO BLOCO
        dim3 blockDims(BLOCK_DEFAULT,1,1);
        // SE A OPCAO DE SHARED MEMORY
        // FOR ATIVADA, DEFINE O TAMANHO
        // DO BLOCO PARA 32
        //if (ct->sharedMemory == 1)
        //    blockDims.x = BLOCK_DIM;
        dim3 gridDims((unsigned int) ceil((double)(linhasIn/blockDims.x)), 1, 1 );

            //Declare GPU pointer
            unsigned char *GPU_input, *GPU_output;
            //Allocate 2D memory on GPU. Also known as Pitch Linear Memory
            size_t gpu_image_pitch = 0;

        // EXECUTA O CUDAMEMCPY
        // ASSINCRONO OU SINCRONO
        if (ct->async == 1) {

            hipMallocPitch<unsigned char>(&GPU_input,&gpu_image_pitch,imageParams->coluna,imageParams->linha);
            hipMallocPitch<unsigned char>(&GPU_output,&gpu_image_pitch,imageParams->coluna,imageParams->linha);

            //Copy data from host to device.
            hipMemcpy2D(GPU_input,gpu_image_pitch,block[numBlock].ppmIn,imageParams->coluna,imageParams->coluna,imageParams->linha,hipMemcpyHostToDevice);

            //Bind the image to the texture. Now the kernel will read the input image through the texture cache.
            //Use tex2D function to read the image
            hipBindTexture2D(NULL,tex8u,GPU_input,imageParams->coluna,imageParams->linha,gpu_image_pitch);

            /*
             * Set the behavior of tex2D for out-of-range image reads.
             * hipAddressModeBorder = Read Zero
             * hipAddressModeClamp  = Read the nearest border pixel
             * We can skip this step. The default mode is Clamp.
             */
            tex8u.addressMode[0] = tex8u.addressMode[1] = hipAddressModeBorder;
        } else
            hipMemcpy( kInput, block[numBlock].ppmIn, linhasIn, hipMemcpyHostToDevice);

        // EXECUTA A FUNCAO SMOOTH NO KERNEL
        // SE A OPCAO DE SHARED MEMORY FOR ATIVADA
        // CHAMA A FUNCAO smoothPPM_SH
        if (ct->async == 1) {
            if (ct->sharedMemory == 1)
                smoothPPM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPPM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        } else {
            if (ct->sharedMemory == 1)
                smoothPPM_SH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPPM_noSH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        }

        // RETORNA A IMAGEM PARA
        // A VARIAVEL DE SAIDA PARA
        // GRAVACAO NO ARQUIVO
        if (ct->async == 1) {
            //Copy the results back to CPU
            hipMemcpy2D(block[numBlock].ppmOut,imageParams->coluna,GPU_output,gpu_image_pitch,imageParams->coluna,imageParams->linha,hipMemcpyDeviceToHost);

            //Release the texture
            hipUnbindTexture(tex8u);
        } else
            hipMemcpy(block[numBlock].ppmOut, kOutput, linhasOut, hipMemcpyDeviceToHost );

        // LIBERA A MEMORIA
        hipFree(kInput);
        hipFree(kOutput);
    }

    // SE A IMAGEM FOR PPM
    if (strcmp(imageParams->tipo, "P5")==0) {
        // VARIAVEL PARA COPIA DA IMAGEM
        // PARA O KERNEL
        PGMPixel* kInput;
        PGMPixel* kOutput;

        // ALOCAR MEMORIA
        hipMalloc( (void**) &kInput, linhasIn);
        hipMalloc( (void**) &kOutput, linhasOut);

        // DEFINICAO DO TAMANHO PADRAO
        // DO BLOCO
        dim3 blockDims(BLOCK_DEFAULT,1,1);
        // SE A OPCAO DE SHARED MEMORY
        // FOR ATIVADA, DEFINE O TAMANHO
        // DO BLOCO PARA 32
        if (ct->sharedMemory == 1)
            blockDims.x = BLOCK_DIM;
        dim3 gridDims((unsigned int) ceil((double)(linhasIn/blockDims.x)), 1, 1 );

        // EXECUTA O CUDAMEMCPY
        // ASSINCRONO OU SINCRONO
        if (ct->async == 1)
            hipMemcpyAsync( kInput, block[numBlock].pgmIn, linhasIn, hipMemcpyHostToDevice, streamSmooth[numBlock] );
        else
            hipMemcpy( kInput, block[numBlock].pgmIn, linhasIn, hipMemcpyHostToDevice);

        // EXECUTA A FUNCAO SMOOTH NO KERNEL
        // SE A OPCAO DE SHARED MEMORY FOR ATIVADA
        // CHAMA A FUNCAO smoothPPM_SH
        if (ct->async == 1) {
            if (ct->sharedMemory == 1)
                smoothPGM_SH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPGM_noSH<<<gridDims, blockDims, 0, streamSmooth[numBlock]>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        } else {
            if (ct->sharedMemory == 1)
                smoothPGM_SH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
            else
                smoothPGM_noSH<<<gridDims, blockDims>>>(kInput, kOutput, imageParams->coluna, imageParams->linha, block[numBlock].li, block[numBlock].lf);
        }

        // RETORNA A IMAGEM PARA
        // A VARIAVEL DE SAIDA PARA
        // GRAVACAO NO ARQUIVO
        if (ct->async == 1)
            hipMemcpyAsync(block[numBlock].pgmOut, kOutput, linhasOut, hipMemcpyDeviceToHost, streamSmooth[numBlock] );
        else
            hipMemcpy(block[numBlock].pgmOut, kOutput, linhasOut, hipMemcpyDeviceToHost );

        // LIBERA A MEMORIA
        hipFree(kInput);
        hipFree(kOutput);
    }

    hipDeviceSynchronize();

    if (ct->debug >= 1)
        printf("Apply Smooth[%d][%s] - li:%d, lf:%d\n",
               numBlock, imageParams->tipo, block[numBlock].li, block[numBlock].lf);

}

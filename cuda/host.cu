#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "host.cuh"
#include "kernel.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void structToArray(PPMImageParams* imageParams, PPMThread* thread,
                   int numThread, unsigned char *cpuIn, int filtro) {

    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<thread[numThread].linhas*imageParams->coluna; t++)
                cpuIn[t] = thread[numThread].ppmIn[t].red;
        if (filtro == 2)
            for(int t=0; t<thread[numThread].linhas*imageParams->coluna; t++)
                cpuIn[t] = thread[numThread].ppmIn[t].green;
        if (filtro == 3)
            for(int t=0; t<thread[numThread].linhas*imageParams->coluna; t++)
                cpuIn[t] = thread[numThread].ppmIn[t].blue;
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<thread[numThread].linhas*imageParams->coluna; t++)
            cpuIn[t] = thread[numThread].pgmIn[t].gray;
    }
}

void arrayToStruct(PPMImageParams* imageParams, PPMThread* thread,
                   int numThread, unsigned char* cpuOut, int filtro) {

    const int linhas = ((thread[numThread].lf-thread[numThread].li)+1)*imageParams->coluna;

    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<linhas; t++)
                thread[numThread].ppmOut[t].red = cpuOut[t];
        if (filtro == 2)
            for(int t=0; t<linhas; t++)
                thread[numThread].ppmOut[t].green = cpuOut[t];
        if (filtro == 3)
            for(int t=0; t<linhas; t++)
                thread[numThread].ppmOut[t].blue = cpuOut[t];
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<linhas; t++)
            thread[numThread].pgmOut[t].gray = cpuOut[t];
    }

}

float applySmoothTexture(initialParams* ct, PPMImageParams* imageParams,
                       PPMThread* thread, int numThread, hipStream_t* streamSmooth, int filtro) {

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    unsigned char *cpuIn, *cpuOut, *gpuIn, *gpuOut;
    cpuIn = (unsigned char *)malloc(thread[numThread].linhasIn);
    cpuOut = (unsigned char *)malloc(thread[numThread].linhasOut);

    int linhas = (thread[numThread].lf-thread[numThread].li)+1;
    const int widthStep = imageParams->coluna;

    structToArray(imageParams, thread, numThread, cpuIn, filtro);

    //Allocate 2D memory on GPU. Also known as Pitch Linear Memory
    size_t gpu_image_pitch = 0;
    gpuErrchk( hipMallocPitch<unsigned char>(&gpuIn,&gpu_image_pitch,imageParams->coluna,thread[numThread].linhas) );
    gpuErrchk( hipMallocPitch<unsigned char>(&gpuOut,&gpu_image_pitch,imageParams->coluna,linhas) );


    //Copy data from host to device.
    gpuErrchk( hipMemcpy2DAsync(gpuIn,gpu_image_pitch,cpuIn,widthStep,imageParams->coluna,thread[numThread].linhas,hipMemcpyHostToDevice, streamSmooth[numThread]) );

    //Bind the image to the texture. Now the kernel will read the input image through the texture cache.
    //Use tex2D function to read the image
    gpuErrchk( hipBindTexture2D(NULL,textureIn,gpuIn,imageParams->coluna,thread[numThread].linhas,gpu_image_pitch) );

    dim3 blockDims(16,16);
    dim3 gridDims;
    gridDims.x = (imageParams->coluna + blockDims.x - 1)/blockDims.x;
    gridDims.y = (thread[numThread].linhas + blockDims.y - 1)/blockDims.y;

    hipEventRecord(start, 0);
    kernelTexture<<<gridDims,blockDims, 0, streamSmooth[numThread]>>>(gpuOut,imageParams->coluna,imageParams->linha,gpu_image_pitch,thread[numThread].lf,thread[numThread].li);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    //Copy the results back to CPU
    gpuErrchk( hipMemcpy2DAsync(cpuOut,widthStep,gpuOut,gpu_image_pitch,imageParams->coluna,linhas,hipMemcpyDeviceToHost, streamSmooth[numThread]) );

    arrayToStruct(imageParams, thread, numThread, cpuOut, filtro);
    exit(1);

    //Release the texture
    hipUnbindTexture(textureIn);

    //Free GPU memory
    hipFree(gpuIn);
    hipFree(gpuOut);
    free(cpuIn);
    free(cpuOut);

    hipEventElapsedTime(&time, start, stop);

    hipDeviceSynchronize();

    if (ct->debug >= 1)
        printf("Apply Smooth[%d][%s] - linhas: %d, li:%d, lf:%d\n",
               numThread, imageParams->tipo, thread[numThread].linhasIn, thread[numThread].li, thread[numThread].lf);

    return time;
}

// FUNCAO __HOST__
// DEFINICAO DOS PARAMETROS DE CHAMADA DO KERNEL
float applySmooth(initialParams* ct, PPMImageParams* imageParams, PPMThread* thread,
                 int numThread, hipStream_t* streamSmooth, int filtro) {

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // DEFINE A QUANTIDADE DE LINHAS DO
    // BLOCO LIDO E DO BLOCO QUE SERA
    // GRAVADO EM DISCO
    unsigned char *cpuIn, *cpuOut, *gpuIn, *gpuOut;
    cpuIn = (unsigned char *)malloc(thread[numThread].linhasIn);
    cpuOut = (unsigned char *)malloc(thread[numThread].linhasOut);

    structToArray(imageParams, thread, numThread, cpuIn, filtro);
    // ALOCAR MEMORIA
    hipMalloc( (void**) &gpuIn, thread[numThread].linhasIn);
    hipMalloc( (void**) &gpuOut, thread[numThread].linhasOut);

    // DEFINICAO DO TAMANHO PADRAO
    // DO BLOCO
    dim3 blockDims(512,1,1);
    dim3 gridDims((unsigned int) ceil((double)(thread[numThread].linhasIn/blockDims.x)), 1, 1 );

    // EXECUTA O CUDAMEMCPY
    // ASSINCRONO
    gpuErrchk( hipMemcpyAsync( gpuIn, cpuIn, thread[numThread].linhasIn, hipMemcpyHostToDevice, streamSmooth[numThread] ) );

    hipEventRecord(start, 0);
    kernel<<<gridDims, blockDims, 0, streamSmooth[numThread]>>>(gpuIn, gpuOut, imageParams->coluna, imageParams->linha, thread[numThread].li, thread[numThread].lf);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    gpuErrchk( hipMemcpyAsync(cpuOut, gpuOut, thread[numThread].linhasOut, hipMemcpyDeviceToHost, streamSmooth[numThread] ) );

    arrayToStruct(imageParams, thread, numThread, cpuOut, filtro);

    // LIBERA A MEMORIA
    hipFree(gpuIn);
    hipFree(gpuOut);
    free(cpuIn);
    free(cpuOut);

    hipEventElapsedTime(&time, start, stop);

    hipDeviceSynchronize();

    if (ct->debug >= 1)
        printf("Apply Smooth[%d][%s] - li:%d, lf:%d\n",
               numThread, imageParams->tipo, thread[numThread].li, thread[numThread].lf);

    return time;
}

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "cuda.cuh"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
// FUNCAO PARA VISUALIZAR AS
// MENSAGENS DE ERRO DO SISTEMA
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// MEMORIA TEXTURE
texture<unsigned char, hipTextureType2D> textureIn;


// FUNCAO KERNEL
// APLICA O SMOOTH UTILIZANDO
// MEMORIA TEXTURE
__global__ void kernelTexture(unsigned char* kOutput,const int coluna, const int linha,
                              const size_t pitch, const int lf, const int li) {

    int x = blockIdx.x * blockDim.x + threadIdx.x; // LINHA
    int y = blockIdx.y * blockDim.y + threadIdx.y; // COLUNA

    // TIRANDO A BORDA DO PROCESSAMENTO
    if ( y > lf-li || x < 2 || x > coluna-2 || (li == 0 && y < 2) || (lf==linha-1 && y > (lf-li)-2) )
        return;

    float sum = 0.0f;
    int cont = 0;

    // SE A IMAGEM NAO FOR O PRIMEIRO BLOCO
    // DEFINE O INICIO PARA DUAS LINHAS ADIANTE
    // PARA NAO PROCESSAR A BORDA
    int inicio = 0;
    if (li != 0)
        inicio = 2;

    for(int l2= -2; l2<=2; l2++) {
        for(int c2=-2; c2<=2; c2++) {
            if(l2 >= 0 && c2 >= 0) {
                sum += tex2D(textureIn, inicio+x+l2, y+c2);
                cont++;
            }
        }
    }

    // ARMAZENDO O RESULTADO
    // NA MEMORIA GLOBAL
    kOutput[y*pitch+x] = static_cast<unsigned char>(sum/cont);
}

// FUNCAO PARA APLICAR SMOOTH
// SEM TEXTURE
__global__ void kernel(unsigned char* kInput, unsigned char* kOutput,
                       const int coluna, const int linha, const int li, const int lf) {

    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

    int c = x % coluna; // COLUNA
    int l = (x-c)/coluna; // LINHA

    // TIRANDO A BORDA DO PROCESSAMENTO
    if ( l > lf-li || c < 2 || c > coluna-2 || (li == 0 && l < 2) || (lf==linha-1 && l > (lf-li)-2) )
        return;

    // APLICANDO O SMOOTH
    float sum = 0.0f;

    for(int l2 = -2; l2 <= 2; ++l2) {
        for(int c2 = -2; c2 <= 2; ++c2) {
            if((c+l2) >= 2 && (c+l2) < coluna-2 && (l+c2) >= -2 && (l+c2) <= lf-li+4) {
                int p = (x + 2*coluna)+(l2*coluna)+c2; // NAO E O PRIMEIRO BLOCO
                if (li == 0)
                    p = x + 2*coluna; // PRIMEIRO BLOCO
                sum += kInput[p];
            }
        }
    }

    // ARMAZENDO O RESULTADO
    // NA MEMORIA GLOBAL
    kOutput[x] = sum/25;
}

// FUNCAO PARA TRANSFORMAR A IMAGEM
// LIDA EM UM ARRAY
// NECESSARIO PARA UTILIZAR MEMORIA TEXTURA
void structToArray(PPMImageParams* imageParams, PPMThread* thread,
                   int numThread, unsigned char *cpuIn, int filtro) {

    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<thread[numThread].linhasIn * imageParams->coluna; t++)
                cpuIn[t] = thread[numThread].ppmIn[t].red;
        if (filtro == 2)
            for(int t=0; t<thread[numThread].linhasIn * imageParams->coluna; t++)
                cpuIn[t] = thread[numThread].ppmIn[t].green;
        if (filtro == 3)
            for(int t=0; t<thread[numThread].linhasIn * imageParams->coluna; t++)
                cpuIn[t] = thread[numThread].ppmIn[t].blue;
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<thread[numThread].linhasIn * imageParams->coluna; t++)
            cpuIn[t] = thread[numThread].pgmIn[t].gray;
    }
}

// FUNCAO PARA TRANSFORMAR A IMAGEM
// DE UM ARRAY PARA UM STRUCT PADRAO DO SISTEMA
// NECESSARIO PARA UTILIZAR MEMORIA TEXTURA
void arrayToStruct(PPMImageParams* imageParams, PPMThread* thread,
                   int numThread, unsigned char* cpuOut, int filtro) {

    if (strcmp(imageParams->tipo, "P6")==0) {
        if (filtro == 1)
            for(int t=0; t<thread[numThread].linhasOut * imageParams->coluna; t++)
                thread[numThread].ppmOut[t].red = cpuOut[t];
        if (filtro == 2)
            for(int t=0; t<thread[numThread].linhasOut * imageParams->coluna; t++)
                thread[numThread].ppmOut[t].green = cpuOut[t];
        if (filtro == 3)
            for(int t=0; t<thread[numThread].linhasOut * imageParams->coluna; t++)
                thread[numThread].ppmOut[t].blue = cpuOut[t];
    }

    if (strcmp(imageParams->tipo, "P5")==0) {
        for(int t=0; t<thread[numThread].linhasOut * imageParams->coluna; t++)
            thread[numThread].pgmOut[t].gray = cpuOut[t];
    }

}

// FUNCAO __HOST__
// PARA CHAMAR O KERNEL COM TEXTURA
float applySmoothTexture(initialParams* ct, PPMImageParams* imageParams,
                       PPMThread* thread, int numThread, hipStream_t* streamSmooth, int filtro) {

    // INICIANDO O TEMPO
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ALOCANDO VARIAVEIS PARA COPIAR
    // E RECEBER A IMAGEM
    unsigned char *cpuIn, *cpuOut, *gpuIn, *gpuOut;
    cpuIn = (unsigned char *)malloc(thread[numThread].linhasIn * imageParams->coluna * sizeof(unsigned char) );
    cpuOut = (unsigned char *)malloc(thread[numThread].linhasOut * imageParams->coluna * sizeof(unsigned char) );

    // CONVERTENDO O PADRAO DO SISTEMA
    // PARA ARRAY
    structToArray(imageParams, thread, numThread, cpuIn, filtro);

    // ALOCANDO VARIAVEIS PARA
    // ENVIAR E RECEBER A IMAGEM
    // PARA O KERNEL
    size_t pitch = 0;
    gpuErrchk( hipMallocPitch<unsigned char>(&gpuIn,&pitch,imageParams->coluna,thread[numThread].linhasIn) );
    gpuErrchk( hipMallocPitch<unsigned char>(&gpuOut,&pitch,imageParams->coluna,thread[numThread].linhasOut) );

    // COPIANDO DADOS DO HOST
    // PARA O DEVICE
    gpuErrchk( hipMemcpy2DAsync(gpuIn,pitch,cpuIn,imageParams->coluna,imageParams->coluna,thread[numThread].linhasIn,hipMemcpyHostToDevice, streamSmooth[numThread]) );

    // ALOCANDO A IMAGEM NA
    // MEMORIA TEXTURA
    gpuErrchk( hipBindTexture2D(NULL,textureIn,gpuIn,imageParams->coluna,thread[numThread].linhasIn,pitch) );

    // DEFININDO O BLOCO
    dim3 blockDims(16,16);
    dim3 gridDims;
    gridDims.x = (imageParams->coluna + blockDims.x - 1)/blockDims.x;
    gridDims.y = (thread[numThread].linhasIn + blockDims.y - 1)/blockDims.y;

    // CHAMANDO O KERNEL
    hipEventRecord(start, 0); // INICIANDO O RELOGIO

    // CHAMANDO O KERNEL
    kernelTexture<<<gridDims,blockDims, 0, streamSmooth[numThread]>>>(gpuOut,imageParams->coluna,imageParams->linha,pitch,thread[numThread].lf,thread[numThread].li);
    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipEventRecord(stop, 0); // PARANDO O RELOGIO
    hipEventSynchronize(stop);

    // COPIANDO OS DADOS
    // DO DEVICE PARA O HOST
    hipMemcpy2DAsync(cpuOut,imageParams->coluna,gpuOut,pitch,imageParams->coluna,thread[numThread].linhasOut,hipMemcpyDeviceToHost, streamSmooth[numThread]);

    // CONVERTENDO O ARRAY RECEBIDO
    // PARA A STRUCT PADRAO DO SISTEMA
    arrayToStruct(imageParams, thread, numThread, cpuOut, filtro);

    // LIBERANDO A TEXTURA
    hipUnbindTexture(textureIn);

    // LIBERANDO MEMORIA
    hipFree(gpuIn);
    hipFree(gpuOut);
    free(cpuIn);
    free(cpuOut);

    // REGISTRANDO O TEMPO
    hipEventElapsedTime(&time, start, stop);

    if (ct->debug >= 1)
        printf("Apply Smooth[%d][%s] - linhas: %d, li:%d, lf:%d\n",
               numThread, imageParams->tipo, thread[numThread].linhasIn, thread[numThread].li, thread[numThread].lf);

    return time;
}

// FUNCAO __HOST__
// DEFINICAO DOS PARAMETROS DE CHAMADA DO KERNEL
float applySmooth(initialParams* ct, PPMImageParams* imageParams, PPMThread* thread,
                 int numThread, hipStream_t* streamSmooth, int filtro) {

    // INICIANDO O TEMPO
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // ALOCANDO VARIAVEIS PARA COPIAR
    // E RECEBER A IMAGEM
    unsigned char *cpuIn, *cpuOut, *gpuIn, *gpuOut;
    cpuIn = (unsigned char *)malloc(thread[numThread].linhasIn * imageParams->coluna * sizeof(unsigned char) );
    cpuOut = (unsigned char *)malloc(thread[numThread].linhasOut * imageParams->coluna * sizeof(unsigned char) );

    // CONVERTENDO O PADRAO DO SISTEMA
    // PARA ARRAY
    structToArray(imageParams, thread, numThread, cpuIn, filtro);

    // ALOCANDO VARIAVEIS PARA
    // ENVIAR E RECEBER A IMAGEM
    // PARA O KERNEL
    hipMalloc( (void**) &gpuIn, thread[numThread].linhasIn * imageParams->coluna);
    hipMalloc( (void**) &gpuOut, thread[numThread].linhasOut * imageParams->coluna);

    // DEFINICAO DO TAMANHO PADRAO
    // DO BLOCO
    dim3 blockDims(512,1,1);
    dim3 gridDims((unsigned int) ceil((double)(thread[numThread].linhasIn * imageParams->coluna/blockDims.x)), 1, 1 );

    // COPIANDO DADOS DO HOST
    // PARA O DEVICE
    hipMemcpyAsync( gpuIn, cpuIn, thread[numThread].linhasIn * imageParams->coluna, hipMemcpyHostToDevice, streamSmooth[numThread] );

    hipEventRecord(start, 0); // INICIANDO O RELOGIO

    // CHAMANDO O KERNEL
    kernel<<<gridDims, blockDims, 0, streamSmooth[numThread]>>>(gpuIn, gpuOut, imageParams->coluna, imageParams->linha, thread[numThread].li, thread[numThread].lf);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );

    hipEventRecord(stop, 0); // PARANDO O RELOGIO
    hipEventSynchronize(stop);

    // COPIANDO OS DADOS
    // DO DEVICE PARA O HOST
    hipMemcpyAsync(cpuOut, gpuOut, thread[numThread].linhasOut * imageParams->coluna, hipMemcpyDeviceToHost, streamSmooth[numThread] );

    // CONVERTENDO O ARRAY RECEBIDO
    // PARA A STRUCT PADRAO DO SISTEMA
    arrayToStruct(imageParams, thread, numThread, cpuOut, filtro);

    // LIBERANDO A MEMORIA
    hipFree(gpuIn);
    hipFree(gpuOut);
    free(cpuIn);
    free(cpuOut);

    hipDeviceSynchronize();

    // REGISTRANDO O TEMPO
    hipEventElapsedTime(&time, start, stop);

    if (ct->debug >= 1)
        printf("Apply Smooth[%d][%s] - li:%d, lf:%d\n",
               numThread, imageParams->tipo, thread[numThread].li, thread[numThread].lf);

    return time;
}



#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>

#include "../cuda/host.cuh"
#include "../common/imagem.cuh"
#include "../common/funcao.cuh"
#include "../common/timer.cuh"

#include "menu.cuh"

int main(int argc, char** argv) {

    // ALOCA MEMORIA PARA OS PARAMETROS
    // DA IMAGEM
    PPMImageParams* imageParams = (PPMImageParams *)malloc(sizeof(PPMImageParams));

    // CARREGA O MENU OU SETA AS OPCOES
    // CASO INSERIDAS NA LINHA DE COMANDO
    initialParams* ct = (initialParams *)calloc(1,sizeof(initialParams));
    ct->DIRIMGIN = "images_in/";  //DIRETORIO DAS IMAGEMS
    ct->DIRIMGOUT = "images_out/"; //DIRETORIO DE SAIDA
    ct->DIRRES = "resultados/"; //GUARDAR OS LOGS
    ct->typeAlg = 'C'; //TIPO DE ALGORITMO, P: PARALELO

    // CARREGA AS OPCOES DO USUARIO
    menu(ct, argc, argv);

    // SETANDO O ARQUIVO DE SAIDA E ENTRADA
    sprintf((char*) &imageParams->fileOut, "%s%s", ct->DIRIMGOUT, ct->filePath);
    sprintf((char*) &imageParams->fileIn, "%s%s", ct->DIRIMGIN, ct->filePath);

    getPPMParameters(ct, imageParams);

    //INFO DO PROCESSO ESCOLHIDO:
    printf("\n\nFile PPM %s\ncoluna: %d\nlinha: %d\nTipo: %s\n", imageParams->fileIn,
                      imageParams->coluna,
                      imageParams->linha,
                      strcmp(imageParams->tipo, "P6")==0?"COLOR":"GRAYSCALE");

    // DEFINE A QUANTIDADE DE LINHAS
    // DA IMAGEM PARA LEITURA E SMOOTH
    int numMaxLinhas = imageParams->linha;

    // SE FOI DEFINIDA A QUANTIDADE DE LINHAS
    // PELO MENU, ALTERAR AQUI
    if (ct->numMaxLinhas > 0)
        numMaxLinhas = ct->numMaxLinhas;
    else {
        int r = 14000000/imageParams->coluna;
        numMaxLinhas = r;
    }

    printf("\nCarga de Trabalho: %d", numMaxLinhas);
    printf("\nMemoria Compartilhada: %s", ct->sharedMemory==1?"Ativado":"Desativado");
    printf("\nMemoria Assincrona: %s\n", ct->async==1?"Ativado":"Desativado");

    timer* tempoC = (timer* )malloc(sizeof(timer)); // RELOGIO APLICACAO
    timer* tempoR = (timer* )malloc(sizeof(timer)); // RELOGIO LEITURA
    timer* tempoS = (timer* )malloc(sizeof(timer)); // RELOGIO SMOOTH
    timer* tempoW = (timer* )malloc(sizeof(timer)); // RELOGIO WRITE

    start_timer(tempoC, "CUDA"); // INICIA O RELOGIO DA APLICACAO

    //GRAVA O CABECALHO DA
    //IMAGEM DE SAIDA
    writePPMHeader(ct, imageParams);

    // CRIA OS CUDA STREAM PARA ASYNC
    hipStream_t streamSmooth[numMaxLinhas];

    if (ct->async == 1)
        for (int i = 0; i < numMaxLinhas; ++i)
            hipStreamCreate(&streamSmooth[i]);

    // ALOCA MEMORIA PARA A QUANTIDADE
    // DE BLOCOS QUE SERAO GERADOS
    int blocks = (imageParams->linha/numMaxLinhas)+1;
    PPMBlock* block = (PPMBlock *)malloc(sizeof(PPMBlock) * blocks);

    // FAZ A DIVISAO DE LINHAS
    // POR BLOCOS
    int i=0;
    while (blocks != 0) {
        blocks = getDivisionBlocks(ct, imageParams, block, 1, i, numMaxLinhas);
        if (blocks == 0)
            continue;
        i++;
    }

    // CRIA UM THREAD PARA CADA DIVISAO
    #pragma omp parallel num_threads(i) shared(i, ct, imageParams, block, t, streamSmooth)
    {
        #pragma omp for
        for(int t=0; t<i; t++) {
            // FAZ A LEITURA DA PARTE DA IMAGEM
            // NO DISCO
            start_timer(tempoR); //INICIA O RELOGIO
            getImageBlocks(ct, imageParams, block,  t);
            stop_timer(tempoR);

            // APLICA O SMOOTH
            start_timer(tempoS); //INICIA O RELOGIO
            applySmooth(ct, imageParams, block, t, streamSmooth);
            stop_timer(tempoS);

            // FAZ A GRAVACAO
            start_timer(tempoW); //INICIA O RELOGIO
            writePPMPixels(ct, imageParams, block, t);
            stop_timer(tempoW);
        }
        #pragma omp barrier
    }

    //PARA O RELOGIO
    show_timer(tempoR, "READ");
    show_timer(tempoS, "SMOOTH");
    show_timer(tempoW, "WRITE");
    stop_timer(tempoC);
    show_timer(tempoC, "CUDA");

    // DESTROI O CUDA STREAM
    if (ct->async == 1)
        for (int i = 0; i < numMaxLinhas; ++i)
            hipStreamDestroy(streamSmooth[i]);

    //ESCREVE NO ARQUIVO DE LOGS
    //writeFile(ct, imageParams, tempo);

    // LIMPAR A MEMORIA
    cleanMemory(imageParams, block, tempoC, ct);

    return 0;

}
